
#include <hip/hip_runtime.h>
#include<iostream>

// CUDA kernel for vector addition
__global__ void vectorAdd(int *a, int *b, int *c, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    c[i] = a[i] + b[i];
  }
}

int main() {
  int n = 10000;
  int *host_vectorA, *host_vectorB, *host_vectorC;
  int *device_vectorA, *device_vectorB, *device_vectorC;
  float elapsed_time;
  hipEvent_t start, stop;
  size_t size = n * sizeof(int);

  // Allocate memory for host vectors
  host_vectorA = new int[size];
  host_vectorB = new int[size];
  host_vectorC = new int[size];

  // Initialize host vectors 
  for(int i=0; i<n; i++) {
    host_vectorA[i] = i;
    host_vectorB[i] = i;
  }

  // Allocate memory for device vectors
  hipMalloc(&device_vectorA, size);
  hipMalloc(&device_vectorB, size);
  hipMalloc(&device_vectorC, size);

  //Copy host vectors to device vectors
  hipMemcpy(device_vectorA, host_vectorA, size, hipMemcpyHostToDevice);
  hipMemcpy(device_vectorB, host_vectorB, size, hipMemcpyHostToDevice);

  // Define block size and grid size
  int blockSize = 256;
  int gridSize = (n + blockSize - 1) / blockSize;

  //Create Cuda events
  hipEventCreate(&start);
  hipEventCreate(&stop);

  //Launch kernel
  hipEventRecord(start);
  vectorAdd<<<gridSize, blockSize>>>(device_vectorA, device_vectorB, device_vectorC, n);
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  //Copy device result vector to host result vector
  hipMemcpy(host_vectorC, device_vectorC, size, hipMemcpyDeviceToHost);

  //Evaluate result
  for(int j=0; j<n; j++) {
    if (host_vectorC[j] != 2*j) {
      std::cout<<"Error Evaluating Result!"<<std::endl;
      break;
    }
  }
  hipEventElapsedTime(&elapsed_time, start, stop);
  std::cout<<"Elapsed Time: "<<elapsed_time<<"ms";

  // Free device memory
  hipFree(device_vectorA); 
  hipFree(device_vectorB);
  hipFree(device_vectorC);

  // Free host memory 
  delete[] host_vectorA;
  delete[] host_vectorB;
  delete[] host_vectorC;

  return 0;
}